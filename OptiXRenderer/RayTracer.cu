#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    // TODO: calculate the color using the Blinn-Phong reflection model

    const float EPS = 1e-4; // prevent self-shadowing

    float3 result = attrib.emission + attrib.ambient;

    // Point Lights
    int plCount = plights.size();
    for (int i = 0; i < plCount; i++)
    {
        PointLight currP = plights[i];
        float3 lightDir = normalize(currP.point - attrib.intersection);

        // curr position light shadow ray
        ShadowPayload shp;
        shp.isVisible = 1; // assume visibility

        // self-shadowing check --> attrib.intersection + EPS * lightDir
        Ray shadow = make_Ray(attrib.intersection, lightDir, 1, EPS, length(currP.point - attrib.intersection));
        rtTrace(root, shadow, shp);
        // now can check shp for visbility

        if (shp.isVisible)
        {
            // Attenutation Factor
            float L = length(currP.point - attrib.intersection); // just distance from point to intersection (not direction)
            float dropoff = currP.attenuation.x + currP.attenuation.y * L + currP.attenuation.z * L * L;
            // dont want to divide by 0
            float attenFactor = (dropoff < 1e-6f) ? 1.0f : (1.0f / dropoff);

            // Diffuse Factor
            float dotnl = dot(attrib.normal, lightDir);
            dotnl = (dotnl < 0.0f) ? 0.0f : dotnl;
            // component wise multiplication
            float3 diffuseFactor = currP.intensity * attrib.diffuse * dotnl;

            // Specular Factor with half-way vector
            float3 h = normalize(attrib.view + lightDir);
            float dotnh = dot(attrib.normal, h);
            dotnh = (dotnh < 0.0f) ? 0.0f : dotnh;
            float3 specFactor = attrib.specular * currP.intensity * powf(dotnh, attrib.shininess);
            
            result += attenFactor * (diffuseFactor + specFactor);
        }
    }

    // Direction Lights
    int dlCount = dlights.size();
    for (int i = 0; i < dlCount; i++)
    {
        DirectionalLight currD = dlights[i];
        float3 lightDir = normalize(currD.direction);

        ShadowPayload shp;
        shp.isVisible = 1; // assume visibilty
        Ray shadow = make_Ray(attrib.intersection, lightDir, 1, EPS, RT_DEFAULT_MAX);
        rtTrace(root, shadow, shp);
        // now check visibility
        if (shp.isVisible)
        {
            // diffuse factor
            float dotnl = dot(attrib.normal, lightDir);
            dotnl = (dotnl < 0.0f) ? 0.0f : dotnl;
            // component wise mul
            float3 diffuseFactor = attrib.diffuse * currD.intensity * dotnl;

            // specular factor
            float3 h = normalize(attrib.view + lightDir); // half way vector
            float dotnh = dot(attrib.normal, h);
            dotnh = (dotnh < 0.0f) ? 0.0f : dotnh;
            float3 specFactor = attrib.specular * currD.intensity * powf(dotnh, attrib.shininess);
            
            result += (diffuseFactor + specFactor);
        }
    }

    // reflections:
    // calc reflection direction: from incidence/-view and normal
    if (payload.maxdepth > 0)
    {
        float3 rDir = reflect(attrib.view * -1, attrib.normal);

        Payload rp;
        rp.radiance = make_float3(0.0f);
        rp.done = 0;
        rp.maxdepth = payload.maxdepth - 1;

        // send out reflection until depth recursively (iterate until depth limit)
        Ray refRay = make_Ray(attrib.intersection, rDir, 0, EPS, RT_DEFAULT_MAX);
        rtTrace(root, refRay, rp);
        
        result += rp.radiance * attrib.specular;
    }

    payload.radiance = result;
}