#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    // ray components
    float3 p0 = ray.origin;
    float3 dir = ray.direction;

    // setup matrix for barycentric equation
    Matrix4x4 p;

    p.setRow(0, make_float4(tri.vert0.x, tri.vert1.x, tri.vert2.x, -1*dir.x));
    p.setRow(1, make_float4(tri.vert0.y, tri.vert1.y, tri.vert2.y, -1*dir.y));
    p.setRow(2, make_float4(tri.vert0.z, tri.vert1.z, tri.vert2.z, -1*dir.z));
    p.setRow(3, make_float4(1.0f, 1.0f, 1.0f, 0.0f));

    
    
    

    // homogenization
    float4 p00 = make_float4(p0, 1.0f);

    // DEBUG: p00 printout
    //rtPrintf("p00: (%f, %f, %f, %f)\n", p00.x, p00.y, p00.z, p00.w);


    Matrix4x4 invp = p.inverse();

    /*
    // DEBUG: p print out 
    for (int i = 0; i < 4; ++i) {
        rtPrintf("Row %d: %f %f %f %f\n", i,
            p[i * 4 + 0],
            p[i * 4 + 1],
            p[i * 4 + 2],
            p[i * 4 + 3]
        );
    }
    rtPrintf("\n\n");
    */

    float4 ans = invp * p00;

    // components
    float lambda1 = ans.x;
    float lambda2 = ans.y;
    float lambda3 = ans.z;
    t = ans.w;
    //rtPrintf("%f\n", t);
    if (lambda1 < 0.0f || lambda2 < 0.0f || lambda3 < 0.0f)
    {
        // DEBUG
        //rtPrintf("Lambda less than 0 fail for triangle %f", primIndex);
        return;
    }
    if (t < 0.0f)
    {
        //rtPrintf("t less than 0 fail for triangle %f", primIndex);
        return;
    }

    // Report intersection (material programs will handle the rest)
    //rtPrintf("t precheck %f", t);
    if (rtPotentialIntersection(t))
    {
        //rtPrintf("t confirmed postcheck %f", t);
        // Pass attributes
        // TODO: assign attribute variables here
        attrib = tri.attrib;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}