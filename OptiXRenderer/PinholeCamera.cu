#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;


rtBuffer<float3, 2> resultBuffer; // used to store the rendered image result

rtDeclareVariable(rtObject, root, , ); // Optix graph

// launchIndex is 2d uint vector with pixel index curr rendering
// rtLaunchIndex gives users current launch index for pixel working on
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here
rtDeclareVariable(float, width, , );
rtDeclareVariable(float, height, , );
rtDeclareVariable(optix::float3, eye, , );
rtDeclareVariable(optix::float3, center, , );
rtDeclareVariable(optix::float3, up, , );
rtDeclareVariable(float, fovy, , );
rtDeclareVariable(float, maxdepth, , );

RT_PROGRAM void generateRays()
{
    
    /*
        // DEBUG: passing camera values from host to device
        rtPrintf("Eye: %f %f %f\n", eye.x, eye.y, eye.z);
        rtPrintf("Center: %f %f %f\n", center.x, center.y, center.z);
        rtPrintf("Up: %f %f %f\n", up.x, up.y, up.z);
        rtPrintf("Fovy: %f\n", fovy);
    */

    float3 result = make_float3(0.f);
     
    // TODO: calculate the ray direction (change the following lines)
    float fixed_fovy = fovy * M_PIf / 180.0f;// in degrees to radians
    float2 offset = make_float2(0.5f); // centered
    float2 currPixel = make_float2(launchIndex) + offset;

    // modifiers
    float alpha = 2.0f * ((currPixel.x) / width) - 1.0f;
    float beta = 1.0f - 2.0f * ((currPixel.y) / height);
    float aspect = (float)width / height;
    float u_mod = alpha * aspect * tan(fixed_fovy / 2.0f);
    float v_mod = beta * tan(fixed_fovy / 2.0f);

    float3 w = normalize(eye - center);
    float3 u = normalize(cross(up, w));
    float3 v = cross(u, w);

    float3 origin = eye; 
    float3 dir = normalize(u_mod*u + v_mod*v - w); 
    float epsilon = 0.001f; 


    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    // 0 for basic ray, 1 for shadow ray
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    payload.maxdepth = maxdepth;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}