#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    // TODO: implement sphere intersection test here
    // apply sphere transform to each ray intersection
    // with transform calc'd t is in sphere space need to transform back to world
    float3 p0 = make_float3(sphere.inv_transform * make_float4(ray.origin, 1.0f));
    float3 dir = normalize(make_float3(sphere.inv_transform * make_float4(ray.direction, 0.0f)));
    // sphere center = origin in sphere space
    float3 c = sphere.center;
    float r = sphere.radius;

    // ray-sphere intersection equation
    float discrim = (dot(dir, (p0 - c)) * dot(dir, (p0 - c))) - (length(p0 - c) * length(p0 - c)) + (r * r);

    if (discrim < 0.0f) // no intersection
        return;
    else if (discrim == 0.0f) // 1 intersection - tangent
    {
        t = dot(-dir, (p0 - c)); // discrim is 0.0f
        if (t <= 0)
            return;
    }
    else // positive discrim, two intersection
    {
        // two possible choices - get smaller positive root
        float t1 = dot(-1 * dir, (p0 - c)) + sqrt(discrim);
        float t2 = dot(-1 * dir, (p0 - c)) - sqrt(discrim);

        if (t1 > 0.0f && t2 > 0.0f)
        {
            t = (t1 > t2) ? t2 : t1;
        }
        else if (t1 > 0.0f)
        {
            t = t1;
        }
        else if (t2 > 0.0f)
        {
            t = t2;
        }
        else
            return;
    }

    float3 sphereHit = (p0 - c) + t * dir;
    float3 worldHit = make_float3(sphere.transform * make_float4(sphereHit, 1.0f));
    float worldT = length(worldHit - ray.origin);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(worldT))
    {
        // Pass attributes

        // TODO: assign attribute variables here
        attrib.intersection = worldHit; // where intersection happens in world
        // sphere center and radius is sphere world need to convert to real world
        float3 normalSphere = normalize(sphereHit - sphere.center); // normal in sphere world
        // use inverse transpose to convert normal in object space to world space

        // get transpose of sphere.inv_transform:
        optix::Matrix4x4 invtrans;

        optix::float4 row0 = sphere.inv_transform.getRow(0);
        optix::float4 row1 = sphere.inv_transform.getRow(1);
        optix::float4 row2 = sphere.inv_transform.getRow(2);
        optix::float4 row3 = sphere.inv_transform.getRow(3);

        invtrans.setRow(0, make_float4(row0.x, row1.x, row2.x, row3.x));
        invtrans.setRow(1, make_float4(row0.y, row1.y, row2.y, row3.y));
        invtrans.setRow(2, make_float4(row0.z, row1.z, row2.z, row3.z));
        invtrans.setRow(3, make_float4(row0.w, row1.w, row2.w, row3.w));

        float4 normalWorld = invtrans * make_float4(normalSphere, 0.0f);
        attrib.normal = normalize(make_float3(normalWorld)); // normal in real world

        attrib.view = normalize(ray.origin - worldHit); // add view ray

        attrib.ambient = sphere.attrib.ambient;
        attrib.diffuse = sphere.attrib.diffuse;
        attrib.shininess = sphere.attrib.shininess;
        attrib.specular = sphere.attrib.specular;
        attrib.emission = sphere.attrib.emission;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // using unit sphere in sphere world need to convert to world space
    // convert all corners of bounding box to cover sphere and then check for min/max
    float3 box[8];
    box[0] = make_float3(-1.0f, -1.0f, -1.0f);
    box[1] = make_float3(1.0f, -1.0f, -1.0f);
    box[2] = make_float3(-1.0f, 1.0f, -1.0f);
    box[3] = make_float3(-1.0f, -1.0f, 1.0f);
    box[4] = make_float3(1.0f, 1.0f, -1.0f);
    box[5] = make_float3(1.0f, -1.0f, 1.0f);
    box[6] = make_float3(-1.0f, 1.0f, 1.0f);
    box[7] = make_float3(1.0f, 1.0f, 1.0f);
    // TODO: implement sphere bouding box

    float3 tbox[8];
    for (int i = 0; i < 8; i++)
    {
        tbox[i] = make_float3(sphere.transform * make_float4(box[i], 1.0f));
    }

    float3 min = tbox[0];
    float3 max = tbox[0];
    for (int i = 1; i < 8; i++)
    {
        min = fminf(min, tbox[i]);
        max = fmaxf(max, tbox[i]);
    }

    result[0] = min.x;
    result[1] = min.y;
    result[2] = min.z;
    result[3] = max.x;
    result[4] = max.y;
    result[5] = max.z;
}