#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    // TODO: implement sphere intersection test here
    float3 p0 = ray.origin;
    float3 dir = ray.direction;
    float3 c = sphere.center;
    float r = sphere.radius;

    float discrim = pow( dot(dir, (p0 - c)) , 2.0f ) - pow(length(p0 - c) , 2.0f) + pow(r, 2.0f);

    if (discrim < 0.0f) // no intersection
        return;
    else if (discrim == 0.0f) // 1 intersection - tangent
    {
        t = dot(-dir, (p0 - c)); // discrim is 0.0f
        if (t <= 0)
            return;
    }
    else // positive discrim, two intersection
    {
        // two possible choices - get smaller positive root
        float t1 = dot(-1 * dir, (p0 - c)) + sqrt(discrim);
        float t2 = dot(-1 * dir, (p0 - c)) - sqrt(discrim);

        if (t1 > 0.0f && t2 > 0.0f)
        {
            t = (t1 > t2) ? t2 : t1;
        }
        else if (t1 > 0.0f)
        {
            t = t1;
        }
        else if (t2 > 0.0f)
        {
            t = t2;
        }
        else
            return;
    }

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes

        // TODO: assign attribute variables here

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}