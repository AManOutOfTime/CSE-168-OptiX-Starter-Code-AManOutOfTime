#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    // TODO: implement sphere intersection test here
    // apply sphere transform to each ray intersection
    // with transform calc'd t is in sphere space need to transform back to world
    float3 p0 = make_float3(sphere.inv_transform * make_float4(ray.origin, 1.0f));
    float3 dir = normalize(make_float3(sphere.inv_transform * make_float4(ray.direction, 0.0f)));
    // sphere center = origin in sphere space
    float3 c = sphere.center;
    float r = sphere.radius;

    float discrim = (dot(dir, (p0 - c)) * dot(dir, (p0 - c))) - (length(p0 - c) * length(p0 - c)) + (r * r);

    if (discrim < 0.0f) // no intersection
        return;
    else if (discrim == 0.0f) // 1 intersection - tangent
    {
        t = dot(-dir, (p0 - c)); // discrim is 0.0f
        if (t <= 0)
            return;
    }
    else // positive discrim, two intersection
    {
        // two possible choices - get smaller positive root
        float t1 = dot(-1 * dir, (p0 - c)) + sqrt(discrim);
        float t2 = dot(-1 * dir, (p0 - c)) - sqrt(discrim);

        if (t1 > 0.0f && t2 > 0.0f)
        {
            t = (t1 > t2) ? t2 : t1;
        }
        else if (t1 > 0.0f)
        {
            t = t1;
        }
        else if (t2 > 0.0f)
        {
            t = t2;
        }
        else
            return;
    }

    float3 sphereHit = (p0 - c) + t * dir;
    float3 worldHit = make_float3(sphere.transform * make_float4(sphereHit, 1.0f));
    float worldT = length(worldHit - ray.origin);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(worldT))
    {
        // Pass attributes

        // TODO: assign attribute variables here
        attrib.ambient = sphere.ambient;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}